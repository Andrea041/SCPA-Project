#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <helper_timer.h>
#include <cmath>
#include <algorithm>

#include "../CUDA_libs/csrTool.h"
#include "../libs/data_structure.h"
#include "../CUDA_libs/csrOperations.h"
#include "../CUDA_libs/cudaCostants.h"

#include <hip/hip_runtime_api.h>

double *y_CPU = nullptr;

/* Funzione per verificare la differenza relativa tra il vettore calcola con GPU e quello con CPU */
void checkDifferences(double *y_h, int matrix_row) {
    double relativeDiff = 0.0f;
    double diff = 0.0f;
    double maxAbs;
    double toleranceRel = 1e-6;  // Tolleranza relativa
    double absTolerance = 1e-7;  // Tolleranza per differenze assolute

    for (int i = 0; i < matrix_row; i++) {
        maxAbs = std::max(std::abs(y_CPU[i]), std::abs(y_h[i]));

        // Se entrambi i valori sono molto piccoli, usiamo una tolleranza relativa
        if (maxAbs < toleranceRel) {
            maxAbs = toleranceRel;  // Imposta un valore minimo per maxAbs
        }

        // Calcolo della differenza assoluta
        double currentDiff = std::abs(y_CPU[i] - y_h[i]);

        // Se la differenza assoluta è sufficientemente piccola, consideriamo i numeri uguali
        if (currentDiff <= absTolerance) {
            relativeDiff = 0.0;
            diff = 0.0;
        } else {
            // Calcoliamo la differenza relativa
            relativeDiff = std::max(relativeDiff, currentDiff / maxAbs);
            diff = std::max(diff, currentDiff);
        }

        /* Si garantisce un errore massimo di precisione nell'ordine di e-7
         * una differenza dell'ordine di 1e-7 è generalmente considerata accettabile per la maggior parte dei calcoli numerici
         * su GPU, quindi non c'è motivo di preoccuparsi a meno che tu non abbia requisiti di precisione estremamente elevata.
         */
        if (relativeDiff > toleranceRel)
            printf("Errore: Il valore di y[%d] calcolato (%.10f) non corrisponde al valore calcolato con CPU (%.10f).\n", i, y_h[i], y_CPU[i]);
    }
}

/* Implementazione del prodotto matrice-vettore seriale su CPU */
matrixPerformance serial_csr_cuda(matrixData *matrix_data_host, double *x_h) {
    int *IRP, *JA;
    double *AS;

    /* Vettore di output del risultato y <- Ax inizializzato su host */
    auto *y_h = static_cast<double *>(malloc(matrix_data_host->M * sizeof(double)));
    if (y_h == nullptr) {
        printf("Errore nell'allocazione della memoria per il vettore di output y\n");
        exit(EXIT_FAILURE);
    }

    /* Conversione in formato CSR */
    convert_to_csr(matrix_data_host->M, matrix_data_host->nz, matrix_data_host->row_indices, matrix_data_host->col_indices, matrix_data_host->values, &IRP, &JA, &AS);

    StopWatchInterface* timer = nullptr;
    sdkCreateTimer(&timer);

    timer->start();
    matvec_csr(matrix_data_host->M, IRP, JA, AS, x_h, y_h);
    timer->stop();

    y_CPU = static_cast<double *>(malloc(matrix_data_host->M * sizeof(double)));
    memcpy(y_CPU, y_h, matrix_data_host->M * sizeof(double));

    matrixPerformance node{};
    node.seconds = timer->getTime()/1000.0f;
    node.flops = 0;
    node.gigaFlops = 0;

    //printf("Time taken by CPU: %f\n", timer->getTime() / 1000.0f);

    free(y_h);
    free(IRP);
    free(JA);
    free(AS);

    return node;
}

/* Implementazione del prodotto matrice-vettore seriale su GPU - v1 */
matrixPerformance parallel_csr_cuda_v1(matrixData *matrix_data_host, double *x_h) {
    int *h_IRP, *h_JA;
    double *h_AS;

    int *d_IRP, *d_JA;
    double *d_AS;
    double *d_y;
    double *d_x;

    /* Vettore di output del risultato y <- Ax inizializzato su CPU */
    auto *y_h = static_cast<double *>(malloc(matrix_data_host->M * sizeof(double)));
    if (y_h == nullptr) {
        printf("Errore nell'allocazione della memoria per il vettore di output y\n");
        exit(EXIT_FAILURE);
    }

    /* Conversione in formato CSR */
    convert_to_csr(matrix_data_host->M, matrix_data_host->nz, matrix_data_host->row_indices, matrix_data_host->col_indices, matrix_data_host->values, &h_IRP, &h_JA, &h_AS);

    /* Allocazione della memoria su GPU */
    checkCudaErrors(hipMalloc((void **) &d_IRP, (matrix_data_host->M + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &d_JA, matrix_data_host->nz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &d_AS, matrix_data_host->nz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &d_y, matrix_data_host->M * sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &d_x, matrix_data_host->N * sizeof(double)));

    /* Copia della memoria da CPU a GPU */
    checkCudaErrors(hipMemcpy(d_IRP, h_IRP, (matrix_data_host->M + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_JA, h_JA, matrix_data_host->nz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_AS, h_AS, matrix_data_host->nz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_x, x_h, matrix_data_host->M * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(d_y, 0, matrix_data_host->M * sizeof(double)));

    StopWatchInterface* timer = nullptr;
    sdkCreateTimer(&timer);
    /* In questo modo ciascun blocco potrà processare una riga in parallelo */
    const dim3 GRID_DIM(matrix_data_host->M);

    timer->start();
    gpuMatVec_csr<<<GRID_DIM, BLOCK_DIM>>>(d_IRP, d_JA, d_AS, d_x, d_y, matrix_data_host->M);
    checkCudaErrors(hipDeviceSynchronize());   //GPU kernel calls are asynchronous: hipDeviceSynchronize() is useful to take the actual execution time on the GPU before timer->stop().
    timer->stop();

    checkCudaErrors(hipMemcpy(y_h, d_y, matrix_data_host->M * sizeof(double), hipMemcpyDeviceToHost));

    /*for (int i = 0; i < matrix_data_host->M; i++) {
        printf("y_h[%d] = %lf\n", i, y_h[i]);
    }*/

    /* Con questa funzione controlliamo se il vettore y ottenuto da GPU è uguale a quello su CPU */
    checkDifferences(y_h , matrix_data_host->M);

    matrixPerformance node{};
    node.seconds = timer->getTime() / 1000.0f;
    node.flops = 0;
    node.gigaFlops = 0;

    printf("Time taken by GPU: %f\n", timer->getTime() / 1000.0f);


    free(y_h);
    free(h_IRP);
    free(h_JA);
    free(h_AS);

    checkCudaErrors(hipFree(d_IRP));
    checkCudaErrors(hipFree(d_JA));
    checkCudaErrors(hipFree(d_AS));
    checkCudaErrors(hipFree(d_y));
    checkCudaErrors(hipFree(d_x));

    return node;
}

/* Implementazione del prodotto matrice-vettore seriale su GPU - v2 */
matrixPerformance parallel_csr_cuda_v2(matrixData *matrix_data_host, double *x_h) {
    int *h_IRP, *h_JA;
    double *h_AS;

    int *d_IRP, *d_JA;
    double *d_AS;
    double *d_y;
    double *d_x;

    /* Vettore di output del risultato y <- Ax inizializzato su CPU */
    auto *y_h = static_cast<double *>(malloc(matrix_data_host->M * sizeof(double)));
    if (y_h == nullptr) {
        printf("Errore nell'allocazione della memoria per il vettore di output y\n");
        exit(EXIT_FAILURE);
    }

    /* Conversione in formato CSR */
    convert_to_csr(matrix_data_host->M, matrix_data_host->nz, matrix_data_host->row_indices, matrix_data_host->col_indices, matrix_data_host->values, &h_IRP, &h_JA, &h_AS);

    /* Allocazione della memoria su GPU */
    checkCudaErrors(hipMalloc((void **) &d_IRP, (matrix_data_host->M + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &d_JA, matrix_data_host->nz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &d_AS, matrix_data_host->nz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &d_y, matrix_data_host->M * sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &d_x, matrix_data_host->N * sizeof(double)));

    /* Copia della memoria da CPU a GPU */
    checkCudaErrors(hipMemcpy(d_IRP, h_IRP, (matrix_data_host->M + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_JA, h_JA, matrix_data_host->nz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_AS, h_AS, matrix_data_host->nz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_x, x_h, matrix_data_host->M * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(d_y, 0, matrix_data_host->M * sizeof(double)));

    /* In questo modo ciascun blocco potrà processare una riga in parallelo */
    const dim3 GRID_DIM(matrix_data_host->M);

    StopWatchInterface* timer = nullptr;
    sdkCreateTimer(&timer);

    timer->start();
    gpuMatVec_csr_sm_seq<<<GRID_DIM, BLOCK_DIM, SHARED_MEM_SIZE>>>(d_IRP, d_JA, d_AS, d_x, d_y, matrix_data_host->M);
    checkCudaErrors(hipDeviceSynchronize());   //GPU kernel calls are asynchronous: hipDeviceSynchronize() is useful to take the actual execution time on the GPU before timer->stop().
    timer->stop();

    checkCudaErrors(hipMemcpy(y_h, d_y, matrix_data_host->M * sizeof(double), hipMemcpyDeviceToHost));

    /* Con questa funzione controlliamo se il vettore y ottenuto da GPU è uguale a quello su CPU */
    checkDifferences(y_h , matrix_data_host->M);

    matrixPerformance node{};
    node.seconds = timer->getTime() / 1000.0f;
    node.flops = 0;
    node.gigaFlops = 0;

    //printf("Time taken by GPU: %f\n", timer->getTime() / 1000.0f);

    free(y_h);
    free(h_IRP);
    free(h_JA);
    free(h_AS);

    checkCudaErrors(hipFree(d_IRP));
    checkCudaErrors(hipFree(d_JA));
    checkCudaErrors(hipFree(d_AS));
    checkCudaErrors(hipFree(d_y));
    checkCudaErrors(hipFree(d_x));

    return node;
}

/* Implementazione del prodotto matrice-vettore seriale su GPU - v3 */
matrixPerformance parallel_csr_cuda_v3(matrixData *matrix_data_host, double *x_h) {
    int *h_IRP, *h_JA;
    double *h_AS;

    int *d_IRP, *d_JA;
    double *d_AS;
    double *d_y;
    double *d_x;

    /* Vettore di output del risultato y <- Ax inizializzato su CPU */
    auto *y_h = static_cast<double *>(malloc(matrix_data_host->M * sizeof(double)));
    if (y_h == nullptr) {
        printf("Errore nell'allocazione della memoria per il vettore di output y\n");
        exit(EXIT_FAILURE);
    }

    /* Conversione in formato CSR */
    convert_to_csr(matrix_data_host->M, matrix_data_host->nz, matrix_data_host->row_indices, matrix_data_host->col_indices, matrix_data_host->values, &h_IRP, &h_JA, &h_AS);

    /* Allocazione della memoria su GPU */
    checkCudaErrors(hipMalloc((void **) &d_IRP, (matrix_data_host->M + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &d_JA, matrix_data_host->nz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &d_AS, matrix_data_host->nz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &d_y, matrix_data_host->M * sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &d_x, matrix_data_host->N * sizeof(double)));

    /* Copia della memoria da CPU a GPU */
    checkCudaErrors(hipMemcpy(d_IRP, h_IRP, (matrix_data_host->M + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_JA, h_JA, matrix_data_host->nz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_AS, h_AS, matrix_data_host->nz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_x, x_h, matrix_data_host->M * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(d_y, 0, matrix_data_host->M * sizeof(double)));

    /* In questo modo ciascun blocco potrà processare una riga in parallelo */
    const dim3 GRID_DIM(matrix_data_host->M);

    StopWatchInterface* timer = nullptr;
    sdkCreateTimer(&timer);

    timer->start();
    gpuMatVec_csr_sm_par<<<GRID_DIM, BLOCK_DIM, SHARED_MEM_SIZE>>>(d_IRP, d_JA, d_AS, d_x, d_y, matrix_data_host->M);
    checkCudaErrors(hipDeviceSynchronize());   //GPU kernel calls are asynchronous: hipDeviceSynchronize() is useful to take the actual execution time on the GPU before timer->stop().
    timer->stop();

    checkCudaErrors(hipMemcpy(y_h, d_y, matrix_data_host->M * sizeof(double), hipMemcpyDeviceToHost));


    /* Con questa funzione controlliamo se il vettore y ottenuto da GPU è uguale a quello su CPU */
    checkDifferences(y_h , matrix_data_host->M);

    matrixPerformance node{};
    node.seconds = timer->getTime() / 1000.0f;
    node.flops = 0;
    node.gigaFlops = 0;

    //printf("Time taken by GPU: %f\n", timer->getTime() / 1000.0f);


    free(y_h);
    free(h_IRP);
    free(h_JA);
    free(h_AS);

    checkCudaErrors(hipFree(d_IRP));
    checkCudaErrors(hipFree(d_JA));
    checkCudaErrors(hipFree(d_AS));
    checkCudaErrors(hipFree(d_y));
    checkCudaErrors(hipFree(d_x));

    return node;
}